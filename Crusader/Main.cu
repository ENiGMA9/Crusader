#include "hip/hip_runtime.h"
﻿#ifndef __CUDA_RUNTIME_H__
#include "hip/hip_runtime.h"                               // This is needed for CUDA calls
#endif

#ifndef __DEVICE_LAUNCH_PARAMETERS_H__
#include ""				    // Again, needed for CUDA calls, don't touch
#endif 

#ifndef _FSTREAM_
#include <fstream>										// Fstream for filehandling, loading, saving, providing settings etc.
#endif

#ifndef _IOSTREAM_
#include <iostream>										// Crusader prints data periodic in the console, I like to use iostreams instead of print functions, also I use hex calls from streams
#endif

#ifndef _INC_TIME
#include <time.h>										// This is intended to be used for benchmark, currently it's use is minimal and not quite fair
#endif 

#ifndef _SSTREAM_
#include <sstream>										// I like working with chars, but sometimes I just need the job done, this is used in the PropertyManager classes
#endif

/* LEGACY_CODE 
#ifndef _STRING_
#include <string>										
#endif
*/

#ifndef _INC_CONIO
#include <conio.h>										// The 'ol conio, I use this to keep console up before closing
#endif

#ifndef PROPERTY_MANAGER_H
#include "PropertyManager/PropertyManager.h"			/* Custom class I created to handle property reading, setting etc. Although initially I didn't mean to
														use classes or cpp in this CUDA project I wanted cleaner main and wanted to do it in a clean OOP manner.*/
														
#endif


// All the values (except THREADSCOUNT) here are further passed to PropertyManager class which edits them according to config.cfg if this exists, otherwise it defaults to these values.
int MAX_WORD_SIZE = 10,									// This specific variable tells Crusader at what word length it should stop, after bruting it
	WORD_BUFFER_SIZE = MAX_WORD_SIZE + 1,				// We set up a word buffer, the reason for the +1 is I sometimes like to keep the first position empty in case I need auxiliary values passed together
	BLOCKS = 256,										// Number of Blocks the CUDA environment should run / cycle
	THREADS = 1024,										// Number of Threads each CUDA block has / cycle
	THREADSCOUNT = BLOCKS*THREADS,						// This variable is just for internal purposes and is not exposed in the config, it's use will become obvious further in the program
	SAVE_CYCLES_COUNT = 250;							/* This is the number of cycles before the saving kicks in, problem with this is cycles completion speed varies depending on GPU power,
															and BLOCKS & THREADS setting, so this need to be replaced soon with a timer*/

/*LEGACY_CODE
unsigned int alphabet_values[39] = {
        '\0', 'a', 'b', 'c', 'd', 'e', 'f', 'g','h', 'i',
        'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 't',
        's', 'u', 'v', 'w', 'x', 'y', 'z', '-','_','0','1',
        '2','3','4','5','6','7','8','9'
    };*/

unsigned int *alphabet_values;				// We set up a pointer which will later become the first adress of our alphabet values array, values number is not specified so this needs to be dynamic

unsigned int word_len_start_size = 1;		// The size from which Crusader starts bruting, this is the default, but it is changed when a file is loaded if it's the case
unsigned int input_hash_count = 1;			// This is used mainly as an iterator starting from 1 for the hashes that are read dynamic
int alphabet_size;							// A value that should store the alphabet size which usually is number_of_characters_in_input_file + 1 for a dead character I need for this algorithm which is on position 0


inline hipError_t CudaBrute(
        unsigned int *return_array,
        unsigned int *stop_flag_end_of_bruting,
        unsigned int *last_bruted_word_size,
        unsigned int *new_word_seed,
        unsigned int *found_matching_hashes_array,
        const unsigned int *input_hash_list_to_match
);

__global__ void protoBruteKernel(
        unsigned int *return_array,
        const unsigned int *seed_word,
        const unsigned int *alphabet,
        const int alphabetSize,
        unsigned int *stop,
        const unsigned int word_len_start_size,
        unsigned int *last_bruted_word_size,
        unsigned int *new_word_seed,
        unsigned int *found_matching_hashes_array,
        const unsigned int *input_hash_list_to_match,
        const unsigned int word_buffer_size,
        const unsigned int max_word_size,
        const unsigned int threadscount
){



	//IMPORTANT: Although I may have missed this in some comments, from beginning until the ending of addition algorithm and transition to the hashing one, the value of the arrays we perform operations on, are actually positions withing a given alphabet, not actual char values.
    unsigned int *cuda_word = new unsigned int[word_buffer_size];                       //This will be the storage for the word assigned to this thread, it represents words as unsigned int arrays because CUDA (7.5) doesn't works well with chars in kernels
    int index = max_word_size;															//This holds the start index for the generation from seed, each Thread generates it's word from a seed specific to each batch + it's thread id
    unsigned int cuda_word_size = 1;													//????????
    unsigned int cuda_word_not_finished = 1;											//???????
    unsigned int thread_unique_index = blockDim.x * blockIdx.x + threadIdx.x;			//We grab the thread unique Id, in CUDA each thread gets an ID within its block starting from 0, because we are working in 1D, we do: blockDimension in X * blockId (each block has it's own id, and thread ids get reset between blocks) * threadLocalID
    unsigned int cuda_word_curr_letter = 0;												//?????
    unsigned int cuda_word_left_overs = thread_unique_index;							//this variable holds if there's anything left to add to the word, it is initialized with the thread unique id so it can generate the word from the seed
    for (int j = index; j >0; j--)
    {
        cuda_word[j] = seed_word[j];													//In this loop we fill our thread word array with the seed word given by the CPU
    }


    do                                                                                 //Here we start looping and doing the addition algorithm to generate the specific word we need to hash and check
    {
        cuda_word[index] += cuda_word_left_overs;										//Add on current position (remember it starts from the rightmost character) all the leftovers
        if (cuda_word[index] >= alphabetSize)											//Do the current position's value exceeds all possible input values (input alphabet size)?
        {
            cuda_word_curr_letter = cuda_word[index];	                                //Save in auxiliary variable our current position value
            cuda_word[index] = (cuda_word_curr_letter % (alphabetSize-1)) + 1;          //The new position value of this specific character will be the modulo of the current position value - 1 because we add afterwards 1 so basically shifting everything to right so we never get position 0 which is reserverd as "\0"
            cuda_word_left_overs = cuda_word_curr_letter / alphabetSize;                //The new leftovers become the div between our saved value and the number of elements in the given alphabet, basically this is digit addition
            
			if (cuda_word_left_overs > 0)												//If there are still left-overs, increase the current word size (we increase to the left)
            {
                cuda_word_size++;
            }
        }
        else                                                                           //If the current position value after adding the leftovers is within input values
        {
            cuda_word_left_overs = 0;													//We have no more leftovers
        }

        if (cuda_word_size >max_word_size || index == 1 || cuda_word_left_overs == 0)	//IF: the new generated word is bigger than the limit size set OR ??????? OR there are no left overs
        {
            cuda_word_not_finished = 0;													//We can say this is not true => We finished generation of word position values
        }

        index--;																		//Decrease the index, it will be relevant if the generation didn't end
    } while (cuda_word_not_finished); //Loop until we decide generation has finished

    if (cuda_word_size<word_len_start_size)                                            
    {
        cuda_word_size = word_len_start_size;
    }


    if (cuda_word_size>max_word_size)                                                   //Is the new generated word bigger than the set limit? If yes we need to signal the bruter should stop at the end of this batch
    {
        return_array[thread_unique_index] = 0;											//We return no value
        *last_bruted_word_size = cuda_word_size;										//?????????We pass the new length we hit !?

        for (int j = max_word_size; j >0; j--)
        {
            new_word_seed[j] = cuda_word[j];                                            //
        }
        *stop = 1;																		//This is the stop flag the CPU thread will look to at the end of each batch, we set this to true so it will know to stop
    }
    else                                                                                //If the word is ok, we can start hashing the word with the real values (values from position values within the alphabet)
    {
        found_matching_hashes_array[thread_unique_index] = 0;                           //We start by setting the return hash as 0 ???????

		//This is where the jenkins hashing algorithm begins, it's adapted to work on our array instead of number decimals
        for (int index = max_word_size - cuda_word_size + 1; index <= max_word_size; index++) //We loop starting from the last generated character (substracting from max size, the current size and advancing by 1)
        {
            if (cuda_word[index]!=0)                                                    //If it's not a 0 (since 0 can't be achieved within the addition algorithm and it's there only at generation, meaning 0 is null) proceess
            {
                found_matching_hashes_array[thread_unique_index] += alphabet[cuda_word[index]];
                found_matching_hashes_array[thread_unique_index] += found_matching_hashes_array[thread_unique_index] << 10;
                found_matching_hashes_array[thread_unique_index] ^= found_matching_hashes_array[thread_unique_index] >> 6;
            }
			else                                                                        //If we did hit a null (awkward?)
			{
                break;
            }
        }
        found_matching_hashes_array[thread_unique_index] += found_matching_hashes_array[thread_unique_index] << 3;
        found_matching_hashes_array[thread_unique_index] ^= found_matching_hashes_array[thread_unique_index] >> 11;
        found_matching_hashes_array[thread_unique_index] += found_matching_hashes_array[thread_unique_index] << 15;

        return_array[thread_unique_index*word_buffer_size] = 0;  //WAT?
    
        for (int index = 1; index <= input_hash_list_to_match[0];index++)
        {
            if (found_matching_hashes_array[thread_unique_index] == input_hash_list_to_match[index])
            {
                return_array[thread_unique_index*word_buffer_size] = cuda_word_size;
                for (int j = max_word_size - cuda_word_size + 1; j <= max_word_size; j++)
                {
                    return_array[j + thread_unique_index*word_buffer_size] = alphabet[cuda_word[j]];
                }
                
                break;
            }
        }
    }

    if (thread_unique_index == threadscount - 1) //Is this the last thread from the batch? If yes:
    {
        *last_bruted_word_size = cuda_word_size;  //The size this thread got to will be transmited to the CPU Thread size count so it knows the new word size
        for (int j = max_word_size - cuda_word_size + 1; j <= max_word_size; j++)
        {
            new_word_seed[j] = cuda_word[j];     //Loop and copy this word position values to the new seed for the CPU
        }
    }
    delete[] cuda_word;                         //Memory free for the dynamic cuda_word, just in case
}
//512 with 1024

int main()
{
	/* We pass variables to the PropertyManager
	TO-DO:
	- Currently syntax in function declaration uses & to retrieve variables by reference, change this behaviour to explicitely send &var adresses from the call so it's easier to read here
	- Initially it was intended to have a Macro that transforms variable names in strings and internally names them in PropertyManager, this needs to be adressed for cleaner, more savvy code
	*/
    PropertyManager::registerProperty("MAX_WORD_SIZE", MAX_WORD_SIZE);
    PropertyManager::registerProperty("BLOCKS", BLOCKS);
    PropertyManager::registerProperty("THREADS", THREADS);
	PropertyManager::registerProperty("SAVE_CYCLES_COUNT", SAVE_CYCLES_COUNT);
   // PropertyManager::parse();					// The parse function actually starts the .cfg reading, until then nothing happens, because of this design, it's possible to implement .cfg hotswap

    WORD_BUFFER_SIZE = MAX_WORD_SIZE + 1;		// Since we read from config we need to update the buffer_size, if hotswap is to be implemented, this needs to go in parse(), maybe have a registerRelation() function
    THREADSCOUNT = BLOCKS*THREADS;				// Same as buffer
	

    std::ifstream save_file_fail_safe("savefile.mace");								// We declare the input savefile name and extension
    std::ifstream input_hashes_file("hashes.txt");									// This is the file from where the hashes are read
    unsigned int file_input_buffer = 0;												// We create a variable (buffer) to help us read, and we make sure it's empty
    unsigned int cpu_last_bruted_word_size = 1;										/* This variable needs to know what last seed the CPU generated,
																					 seed is needed because CUDA should not modify input array => After a cycle, CPU needs to calculate the next seed alone by jumping by THREADSCOUNT units*/
	unsigned int *input_hash_list_to_match;											// This is the start adress pointer of the array which will hold all the hashes read from file
    //LEGACY CODE unsigned int *cpu_word_seed = new unsigned int[WORD_BUFFER_SIZE];				// We create an array big enough so the CPU can generate until the last seed needed
    unsigned int *cuda_results = new unsigned int[THREADSCOUNT*WORD_BUFFER_SIZE];	// This is the array in which the results come from CUDA, it's size is determined by formula: For each thread run by CUDA it needs to have the full buffer size
    unsigned int *stop_flag_end_of_bruting = new unsigned int[1];					// Simple variable I use to tell the CPU algorithm to stop starting cycles since CUDA finished the wordsize in it's threads
    unsigned int *last_bruted_word_size = new unsigned int[1];						// In this variable we read values from the GPU, if the GPU value is bigger than the current value of variable it means the Threads finished a word size and moved onto the next
    unsigned int *new_word_seed = new unsigned int[WORD_BUFFER_SIZE];				/////////// WTF!?
    unsigned int *found_matching_hashes_array = new unsigned int[THREADSCOUNT];
    int index = 0;
    int days, hours, minutes, seconds;
    int completed_cuda_cycles_count = 0;


	try {																				//Just a usual handling of input/output with a try and catch statement
		std::ifstream alphabet_file_input("alphabet.txt");								// Declaring the alphabet input file which currently is handled with the static naming
		if (alphabet_file_input.good()) {												// Alphabet file exists and is not in use?
			std::string alphabet_buffer;												// We need a variable to read from file
			std::getline(alphabet_file_input, alphabet_buffer);							// Read the alphabet file into the buffer

			alphabet_size = alphabet_buffer.length()+1;									/* Because of the way the algorithm in cuda kernel currently works,
																						 we need to have a null value on 1st element (position 0) of the alphabet array*/

			// LEGACY_CODE alphabet_values = new unsigned int[alphabet_size+1];		 
			alphabet_values = new unsigned int[alphabet_size];						    // Declare the alphabet array values with the modified size from before. 

			for (int index = 1;index <= alphabet_buffer.length();index++)
			{
				alphabet_values[index] = (int)alphabet_buffer[index-1];					// Copy each char/int into the array values, there's a typecast to ensure data type
			}
			alphabet_values[0] = (int)'\0';												// This is the null on 0 position I commented above, needed for th

		}
		else                                                                           
		{
			/*If the alphabet file is either in use or doesn't exist, we throw with an int,
			/ not the most beautiful throw statement but it does the job*/

			// LEGACY_CODE std::cout << "Alphabet could not be loaded, check if file 'alphabet.txt' exists or is not in use.";
			throw 0;
		}
	}
	catch (int e)											
	{

		/* If the try/catch threw, we need to stop, although it is a try/catch and it's first purpose is to avoid crashing,
		currently Crusader doesn't handle a missing alphabet although it's easily fixable*/

		std::cout << "Alphabet could not be loaded, check if file 'alphabet.txt' exists or is not in use. Crusader needs to be re-opened."; // Print something for the user
		_getch(); // Make sure console stays there for the user to read the print
		return 0; // Exit the program (this return is in main)
	}

    if (save_file_fail_safe.good())
    {
		
        std::cout << "Savefile detected, loaded progress!\n";
		unsigned int buffer;
		/*std::cout << "Loaded word: ";
		save_file_fail_safe >> word_len_start_size;
        for (index = 1; index <= word_len_start_size; index++)
        {
            save_file_fail_safe >> file_input_buffer;
            new_word_seed [MAX_WORD_SIZE - word_len_start_size + index] = file_input_buffer;
            std::cout << (char)alphabet_values[file_input_buffer];
        }
        std::cout << " with size " << word_len_start_size<<"\n";*/
		save_file_fail_safe >> buffer;
		if (buffer == 1)
		{
			std::cout << "This savefile is from a finished bruting, Crusader will start from scratch.\n";
			for (index = MAX_WORD_SIZE; index >= 0; index--)
			{
				*(new_word_seed + index) = 0;
			}
			*(new_word_seed + MAX_WORD_SIZE) = 1;
		}
		else
		{
			int alphabet_start = save_file_fail_safe.tellg();
			char alphabet_difference = 0;
			std::string alphabet_test;
			save_file_fail_safe >> alphabet_test;
			for (int i = 1; i <= alphabet_size; i++)
			{
				if (alphabet_test.at(i-1) != (char)alphabet_values[i])
				{
					alphabet_difference = 1;
				}
			}
			if (alphabet_difference == 1)
			{
				"Alphabet discrepancy detected. Alphabet used for savefile is different than current alphabet. While this is not a big problem and bruting could continue after safety checks, because at this momment there is no direct interaction with these systems Crusader will not continue until current alphabet is changed to match savefile or savefile is deleted. NOTE: Please do not manually alter the alphabet in savefile until the new alphabet is same size or bigger than previous, also this is not recommended because you are skipping new results by doing that, until you know exactly what you are doing.";
				_getch();
				return 0;
			}









			//THERE IS SOMETHING MISSING HERE - NEED TO FIX.
			/*else
			{
				save_file_fail_safe >> buffer;
				if (buffer
			}*/
			
		}
    }
    else              //This happens if the fail is not available, either being missing either it's in use
    {
        std::cout<<"Savefile could not be found or is in use, starting from scratch\n";        //Let the user know
        for (index = MAX_WORD_SIZE; index >= 0; index--)                                       //Since we start from scratch we initialize everything with 0 -the null value, not first element- with the size of the max of current config.
        {
            *(new_word_seed + index) = 0;
        }
        *(new_word_seed + MAX_WORD_SIZE) = 1;													//We let last character to be first index of the alphabet because Threads start from 0 and we use their indexes for generation
    }
    save_file_fail_safe.close();                                                               //We don't need the input save fail anymore

    if (input_hashes_file.good()){																//Does the input hashes list exists or is not in use?

		unsigned int counter = 0;																//Need to keep track how many of these we have
		while (input_hashes_file >> std::hex >> file_input_buffer)								//Count ever one of them
		{
			counter++;
		}


		input_hash_list_to_match = new unsigned int[counter+1];                                 //Allocate new memory with the size of the input list
		input_hashes_file.clear();																//We clear any operation remnants we did on the file
		input_hashes_file.seekg(0, std::ios::beg);                                              //We position ourselves at the start
        while (input_hashes_file >> std::hex >> file_input_buffer)								//Begin reading the hashes in the new list
        {
            input_hash_list_to_match[input_hash_count] = file_input_buffer;
			input_hash_count++;
        }
        input_hash_list_to_match[0] = counter;
    }
    else
    {
        std::cout << "Input hash list (hashes.txt) not found, check if it's valid. Crusader will start anyway";
    }


    for (index = 0; index < THREADSCOUNT; index++)			//We iterate on the array of results of each Cycle (which is determined by THREADSCOUNT aka THREADS*BLOCKS)
    {
        *(cuda_results + index) = 0;						//Set them to 0, 0 will be null, any other value returned will be a positive match and will be outputted
    }
    clock_t clockt_start_timestamp, clockt_end_timestamp;  //Init vars for time control
	clockt_start_timestamp = clock();                      // Start clockin'

    do{			//This is where a cycle begins
        hipError_t cudaStatus = CudaBrute(cuda_results, stop_flag_end_of_bruting,last_bruted_word_size,new_word_seed,found_matching_hashes_array,input_hash_list_to_match); //We init a hipError_t variable which is a cuda default type for errors and returned from kernels


        if (cudaStatus != hipSuccess) {			//We check if the call failed
            fprintf(stderr, "CudaBrute failed!");
            return 1;
        }


        for (index = 0; index < THREADSCOUNT; index++)   //Begin the sweep for results
        {
            if (cuda_results[index*WORD_BUFFER_SIZE] != 0){  //Is there a result?
                std::ostringstream fileNameStream("");		//Create an empty outputstringstream to use as hex name fo the file
                fileNameStream << std::hex << found_matching_hashes_array[index] << ".txt"; //Concatenate the name for the target file
                std::ofstream output(fileNameStream.str(), std::ios::app);	//Open with append argument the file for which the hash is
                std::cout << "String found: ";
                for (int j = MAX_WORD_SIZE - cuda_results[index*WORD_BUFFER_SIZE]+1; j <= MAX_WORD_SIZE; j++)
                {
                    std::cout << (char)cuda_results[j + index*WORD_BUFFER_SIZE];
                    output << (char)cuda_results[j + index*WORD_BUFFER_SIZE];
                }
                std:: cout << "\n";
                output << "\n";
                output.close();
            }
        }
    
        cpu_last_bruted_word_size = word_len_start_size;
        word_len_start_size = *last_bruted_word_size;
        if (word_len_start_size != cpu_last_bruted_word_size){
    
            clockt_end_timestamp = clock();
            seconds = (float)(clockt_end_timestamp - clockt_start_timestamp) / CLOCKS_PER_SEC;
            minutes = seconds / 60;
            seconds %= 60;
            hours = minutes / 60;
            minutes %= 60;
            days = hours / 24;
            hours %= 24;

            std::cout << "Word size "<<cpu_last_bruted_word_size<< " finished in " << days<<"d "<<hours<<"h "<<minutes<<"m "<<seconds << "s\n"; clockt_start_timestamp = clock();
        }
        if (completed_cuda_cycles_count > SAVE_CYCLES_COUNT){
            std::ofstream output("savefile.mace");
            std::cout << "Saved new start point: ";
			output << cpu_last_bruted_word_size<<" ";
            for (int j = 0; j < MAX_WORD_SIZE; j++)
				//FIIIIX SAVING
            {
                std::cout << (char)alphabet_values[new_word_seed[j]];
                output << new_word_seed[j]<<" ";
                completed_cuda_cycles_count = 0;
            }
            std::cout << "\n";
            output.close();
        }
        completed_cuda_cycles_count++;
    } while (*stop_flag_end_of_bruting != 1);

    std::ofstream output("FINISHED");
    output.close();
    
    clockt_end_timestamp = clock();
    seconds = (clockt_end_timestamp - clockt_start_timestamp) / CLOCKS_PER_SEC;


	//LEGACY CODE  delete[] cpu_word_seed;
	//SaveManager::cleanup();
    delete[] cuda_results;
    delete stop_flag_end_of_bruting;
    delete last_bruted_word_size;
    delete[] new_word_seed;
    delete[] input_hash_list_to_match;
    delete[] found_matching_hashes_array;
	delete[] alphabet_values;
    return 0;



}

inline hipError_t CudaBrute(
        unsigned int *return_array,
        unsigned int *stop_flag_end_of_bruting,
        unsigned int *last_bruted_word_size,
        unsigned int *new_word_seed,
        unsigned int *found_matching_hashes_array,
        const unsigned int *input_hash_list_to_match
){
    unsigned int *gpu_return_array = 0;
    unsigned int *gpu_new_word_seed = 0;
    unsigned int *gpu_stop_flag_end_of_bruting = 0;
    unsigned int *gpu_alphabet_values = 0;
    unsigned int *gpu_last_bruted_word_size = 0;
    unsigned int *gpu_last_word_bruted = 0;
    unsigned int *gpu_found_matching_hashes_array = 0;
    unsigned int *gpu_input_hash_list = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input_hashes_file, one output)
    cudaStatus = hipMalloc((void**)&gpu_return_array, THREADSCOUNT * WORD_BUFFER_SIZE * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&gpu_last_bruted_word_size, sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&gpu_last_word_bruted, WORD_BUFFER_SIZE * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&gpu_new_word_seed, WORD_BUFFER_SIZE * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&gpu_alphabet_values, (alphabet_size+1) * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&gpu_stop_flag_end_of_bruting, sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&gpu_found_matching_hashes_array, THREADSCOUNT * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&gpu_input_hash_list, (input_hash_list_to_match[0]+1) *sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input_hashes_file vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(gpu_new_word_seed, new_word_seed, WORD_BUFFER_SIZE * sizeof(unsigned int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(gpu_alphabet_values, alphabet_values, (alphabet_size+1) * sizeof(unsigned int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(gpu_input_hash_list, input_hash_list_to_match, (input_hash_list_to_match[0]+1) * sizeof(unsigned int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    // Launch a kernel on the GPU with one thread for each element.
    protoBruteKernel << <BLOCKS, THREADS >> >(gpu_return_array, gpu_new_word_seed, gpu_alphabet_values, alphabet_size, gpu_stop_flag_end_of_bruting, word_len_start_size, gpu_last_bruted_word_size, gpu_last_word_bruted, gpu_found_matching_hashes_array, gpu_input_hash_list, WORD_BUFFER_SIZE, MAX_WORD_SIZE, THREADSCOUNT);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "bruteKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching bruteKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.

    cudaStatus = hipMemcpy(return_array, gpu_return_array, THREADSCOUNT * WORD_BUFFER_SIZE * sizeof(unsigned int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(found_matching_hashes_array, gpu_found_matching_hashes_array, THREADSCOUNT * sizeof(unsigned int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(last_bruted_word_size, gpu_last_bruted_word_size, sizeof(unsigned int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(new_word_seed, gpu_last_word_bruted, WORD_BUFFER_SIZE * sizeof(unsigned int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(stop_flag_end_of_bruting, gpu_stop_flag_end_of_bruting, sizeof(unsigned int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(gpu_new_word_seed);
    hipFree(gpu_return_array);
    hipFree(gpu_stop_flag_end_of_bruting);
    hipFree(gpu_last_bruted_word_size);
    hipFree(gpu_alphabet_values);
    hipFree(gpu_last_word_bruted);
    hipFree(gpu_input_hash_list);
    hipFree(gpu_found_matching_hashes_array);
    return cudaStatus;
}
